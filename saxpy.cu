
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>

#define N 100

__global__ void inc(int *a) {
    int i = blockIdx.x;
    if (i<N) {
        a[i]= 1;
    }
}

int main() {
    int ha[N], *da;
    hipMalloc((void **)&da, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    inc<<<N, 1>>>(da);
    hipMemcpy(ha, da, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; ++i) {
        
        std::cout<<ha[i]<<" ";
    }
    
    hipFree(da);
    return 0;
}