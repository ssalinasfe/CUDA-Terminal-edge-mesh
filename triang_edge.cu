#include "hip/hip_runtime.h"
/*Triangulation operations to work with edges instead of triangles*/


#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "consts.h"




//Calcula la distancia entre los puntos |x - y|
__device__ double dist(double x0, double y0, double x1, double y1)
{
	return sqrt(pow(x0 - x1, 2.0) + pow(y0 - y1, 2.0));
}

__device__ int Equality(double a, double b, double epsilon)
{
  return fabs(a - b) < epsilon;
}

//verifica si a es mayor a b de acuerdo a un epsion
__device__ int GreaterEqualthan(double a, double b, double epsilon){
	return Equality(a,b,epsilon) || a > b;
}

/* max_edge_index
 * 
 * Retorna el índice k de la arista máxima de un triángulo i, 
 * descrito por los puntos p0p1p2. Será 0 si p0p1 es máxima.
 * Será 1 si p1p2 lo es. Será 2 si p2p0 lo es.
 * */
__device__ int max_edge_index(int i, double *r, int *p){
     double l0;
     double l1;
     double l2;
     
     int p0;
     int p1;
     int p2;
     
     p0 = p[3*i + 0];
     p1 = p[3*i + 1];
     p2 = p[3*i + 2];
     
     l0 = dist(r[2*p0 + 0], r[2*p0 + 1], r[2*p1 + 0], r[2*p1 + 1]);
     l1 = dist(r[2*p1 + 0], r[2*p1 + 1], r[2*p2 + 0], r[2*p2 + 1]);
     l2 = dist(r[2*p2 + 0], r[2*p2 + 1], r[2*p0 + 0], r[2*p0 + 1]);

     double epsion = 0.001f;
 
     if( (GreaterEqualthan(l0,l1,epsion) && GreaterEqualthan(l1,l2,epsion)) || ( GreaterEqualthan(l0,l2,epsion) && GreaterEqualthan(l2,l1,epsion)))
     {
         return 0;
     }
     else if((GreaterEqualthan(l1,l0,epsion) && GreaterEqualthan(l0,l2,epsion)) || ( GreaterEqualthan(l1,l2,epsion) && GreaterEqualthan(l2,l0,epsion)))
     {
         return 1;
     }
     else
     {
         return 2;
     }
} 


/* same_edge
 * 
 * Indica para las aristas {u,v} y {w,x} si son iguales o no.
 * */
 
 __device__ int same_edge(int u, int v, int w, int x)
 {
     return (u == w && v == x) || (u == x && v == w);
 }


/* get_edge_index
 * 
 * Entrega el índice de la arista {u,v} respecto del triángulo i.
 * */

 __device__  int get_edge_index(int u, int v, int i, int *p)
 {
     int p0;
     int p1;
     int p2;
     
     p0 = p[3*i + 0];
     p1 = p[3*i + 1];
     p2 = p[3*i + 2];
     
     if(same_edge(u, v, p0, p1))
     {
         return 0;
     }
     else if(same_edge(u, v, p1, p2))
     {
         return 1;
     }
     else if(same_edge(u, v, p2, p0))
     {
         return 2;
     }
    
     /*
     else
     {
         fprintf(stderr, "%s:%d:%s() ** ERROR ** get_edge_index: Arista {%d,%d} no pertenece al triángulo %d.\n", __FILE__,  __LINE__, __func__, u, v, i);
         exit(EXIT_FAILURE);
     }*/
 }

/* is_nomax_nomax
 * 
 * Indica si la arista compartida entre los triángulos i y j
 * es nomáx-nomáx.
 * */

 __device__ int is_nomax_nomax(int i, int j, int *p, int *max)
 {
     int p0i;
     int p1i;
     int p2i;
     int p0j;
     int p1j;
     int p2j;
     
     p0i = p[3*i + 0];
     p1i = p[3*i + 1];
     p2i = p[3*i + 2];
     
     p0j = p[3*j + 0];
     p1j = p[3*j + 1];
     p2j = p[3*j + 2];
     
     int ij;
     int ii;
     
     if(same_edge(p0i, p1i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 2;
     }
     else if(same_edge(p0i, p1i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 2;
     }
     else if(same_edge(p0i, p1i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 2;
     }
     /*
     else
     {
         fprintf(stderr, "** ERROR ** is_nomax_nomax: Problema insperado para triángulos %d y %d.\n", i, j);
         exit(EXIT_FAILURE);
     }*/
     
     return (ij != max[j]) && (ii != max[i]);
 }

/* is_max_max
 * 
 * Indica si la arista compartida entre los triángulos i y j
 * es máx-máx.
 * */

 __device__ int is_max_max(int i, int j, int *p, int *max)
 {
     int p0i;
     int p1i;
     int p2i;
     
     int p0j;
     int p1j;
     int p2j;
     
     p0i = p[3*i + 0];
     p1i = p[3*i + 1];
     p2i = p[3*i + 2];
     
     p0j = p[3*j + 0];
     p1j = p[3*j + 1];
     p2j = p[3*j + 2];
     
     int ij;
     int ii;
     
     if(same_edge(p0i, p1i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 2;
     }
     else if(same_edge(p0i, p1i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 2;
     }
     else if(same_edge(p0i, p1i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 2;
     }
   
     
     return (ij == max[j]) && (ii == max[i]);
 }
 




/* edge_belongs_to
 * 
 * Indica si arista {k,l} pertenece al triángulo i.
 * */

 __device__ int edge_belongs_to(int k, int l, int i, int *p)
 {
     return same_edge(k, l, p[3*i + 0], p[3*i + 1])
                     || same_edge(k, l, p[3*i + 1], p[3*i + 2])
                     || same_edge(k, l, p[3*i + 2], p[3*i + 0]);
 }



/* Given one triangle i, return the edge index that containts u and v*/
__device__ int get_shared_edge(int i, int u, int v, int *p){
	int j, ind1,ind2;
	for(j = 0; j < 3; j++){
		ind1 = 3*i + j;
		ind2 = 3*i + (j+1)%3;
		//debug_print("%d %d %d %d %d\n", ind1, ind2, ind3, (p[ind1] == u || p[ind2] == u), (p[ind1] == v || p[ind2] == v));
		if( (p[ind1] == u || p[ind2] == u) && (p[ind1] == v || p[ind2] == v))
			return (j+2)%3;
	}
	return 0;
}

__global__ void label_longest_edges(int *cu_max, double *cu_r, int *cu_triangles, int tnumber)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < tnumber)
    {
        cu_max[i] = max_edge_index(i,cu_r, cu_triangles);
    }

}

__global__ void get_seeds(int *cu_max, int *cu_triangles, int *cu_adj, int *cu_seed, int enumber)
{
    int N = blockDim.x * blockIdx.x + threadIdx.x;
    int i = floorf(N/3);
    int j = N - 3*i;
    if(N < enumber)
    {
        if(cu_adj[N] != -1 && is_max_max(i, cu_adj[N], cu_triangles, cu_max) == TRUE)
        {
            if(cu_adj[N] < i){ //si hay dos triangulos a ser semilla se elige el con menor indice
                cu_seed[i] = TRUE;
                
            }
        }
        //esto se puede optimizar, mezclaro con la operación de arriba
        if (cu_adj[N] == -1 && cu_max[i] == (j+1)%3){ //si es terminal-boder edge
            cu_seed[i] = TRUE;
        }
    }
}

__global__ void label_frontier_edges(int *cu_max, int *cu_triangles, int *cu_adj, int enumber)
{
    int N = blockDim.x * blockIdx.x + threadIdx.x;
    int i = floorf(N/3);
    if(N < enumber)
    {
        //cu_disconnect[N] = (cu_adj[N] != -1) && is_nomax_nomax(i, cu_adj[N], cu_triangles, cu_max);
        cu_adj[N] = ((cu_adj[N] < 0) || is_nomax_nomax(i, cu_adj[N], cu_triangles, cu_max)) ? -1 : cu_adj[N];
    }

}

__global__ void disconnect_edges(int *cu_adj, int* cu_disconnect, int enumber){
    int N = blockDim.x * blockIdx.x + threadIdx.x;
    if(N < enumber)
    {
        cu_adj[N] = (cu_disconnect[N] == TRUE) ? NO_ADJ : cu_adj[N];
    }        
}


__global__ void initialize_memory(int *cu_seed, int* cu_trivertex, int* cu_triangles, int tnumber){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    if(i < tnumber){
    cu_seed[i] = FALSE;  
    }
}
