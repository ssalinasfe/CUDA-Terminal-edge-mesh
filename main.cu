#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>   
#include <stdlib.h>     /* exit, EXIT_FAILURE */
#include "detri2.h"
#include "polymesh.h"
#include <vector> 
#include <chrono>
#include <iomanip>
#include <cstdlib>
#include <algorithm>    // std::min


#include "consts.h"


//cuda
#include "io.cuh"

#include "triang_edge.cuh"
#include "polygon.cuh"
#include "BET_elimination.cuh"


#ifdef DEBUG
#define DEBUG_TEST 1
#else
#define DEBUG_TEST 0
#endif

#define debug_block(fmt) do { if (DEBUG_TEST){ fmt }} while (0)
#define debug_print(fmt, ...) do { if (DEBUG_TEST) fprintf(stderr, "%s:%d:%s(): " fmt, __FILE__, __LINE__, __func__, __VA_ARGS__); } while (0)
#define debug_msg(fmt) do { if (DEBUG_TEST) fprintf(stderr, "%s:%d:%s(): " fmt, __FILE__,  __LINE__, __func__); } while (0)


int main(int argc, char* argv[])
{

    int nparam = 3;
    //char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("test.node")};
	//char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("506randompoints.node")};
	//char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("506equilateral.node")};
    int print_triangles = 0;
    char* ppath;
    //char* ppath = const_cast<char*> ("test");
    //TMesh *Tr = new TMesh(nparam, params);    
	auto tb_delaunay = std::chrono::high_resolution_clock::now();
	TMesh *Tr = new TMesh(argc, argv);    	
	auto te_delaunay = std::chrono::high_resolution_clock::now();
    //Tr->print();
    
	int tnumber, pnumber, i,j;
	double *r;
	int *triangles;
	int *adj;
    int *seed;
	int *max;
	int *mesh;
	int *disconnect;
	int *ind_poly;
	
	
    tnumber = Tr->tnumber;
    pnumber = Tr->pnumber;

	max = (int *)malloc(tnumber*sizeof(int));
	disconnect = (int *)malloc(3*tnumber*sizeof(int));
	seed = (int *)malloc(tnumber*sizeof(int));
    r = (double *)malloc(2*tnumber*sizeof(double));
    adj =(int *)malloc(3*tnumber*sizeof(int));
    triangles = (int *)malloc(3*tnumber*sizeof(int));
	mesh = (int *)malloc(3*tnumber*sizeof(int));
	ind_poly = (int *)malloc(3*tnumber*sizeof(int));
	

	//Cuda functions
    // Initialize device pointers.
    double *cu_r;
	int *cu_triangles;
	int *cu_adj;
    int *cu_seed;
	int *cu_max;
	int *cu_disconnect;
	int *cu_mesh;
	int *cu_ind_poly;

	// Allocate device memory.
	hipMalloc((void**) &cu_max, tnumber*sizeof(int));
	hipMalloc((void**) &cu_seed, tnumber*sizeof(int));
	hipMalloc((void**) &cu_disconnect, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_r, 2*tnumber*sizeof(double));
	hipMalloc((void**) &cu_triangles, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_adj, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_mesh, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_ind_poly, tnumber*sizeof(int));

	/* Llamada a detr2 */
	{


    int idx =0;
    //copiar arreglo de vertices
    //std::cout<<"pnumber "<<pnumber<<std::endl;
    for (i = 0; i < Tr->trimesh->ct_in_vrts; i++) {
        if (!Tr->trimesh->io_keep_unused) { // no -IJ
            if (Tr->trimesh->in_vrts[i].typ == UNUSEDVERTEX) continue;
        }
        r[2*i + 0]= Tr->trimesh->in_vrts[i].crd[0];
        r[2*i + 1]= Tr->trimesh->in_vrts[i].crd[1];
        //std::cout<<idx<<" ("<<r[2*i + 0]<<", "<<r[2*i + 1]<<") "<<std::endl;
        Tr->trimesh->in_vrts[i].idx = idx;
        idx++;
    }
    idx = 0;
    for (int i = 0; i < Tr->trimesh->tr_tris->used_items; i++) {
        detri2::Triang* tri = (detri2::Triang *) Tr->trimesh->tr_tris->get(i);
        if (tri->is_deleted()) continue;
        if (tri->is_hulltri()) {
            tri->idx = -1;
        } else {
            tri->idx = idx;
            idx++;
        }
    }

    //std::cout<<"tnumber: "<<Tr->trimesh->tr_tris->objects - Tr->trimesh->ct_hullsize<<std::endl;
    idx = 0;
    for (int i = 0; i < Tr->trimesh->tr_tris->used_items; i++)
    {
        
        detri2::Triang* tri = (detri2::Triang *) Tr->trimesh->tr_tris->get(i);
        if (tri->is_deleted() || tri->is_hulltri()) continue;
        triangles[3*idx+0] = tri->vrt[0]->idx;
        triangles[3*idx+1] = tri->vrt[1]->idx;
        triangles[3*idx+2] = tri->vrt[2]->idx;
        adj[3*idx+ 0] = tri->nei[0].tri->idx;
        adj[3*idx+ 1] = tri->nei[1].tri->idx;
        adj[3*idx+ 2] = tri->nei[2].tri->idx;
        //std::cout<<idx<<" | "<<triangles[3*idx+0]<<" "<<triangles[3*idx+1]<<" "<<triangles[3*idx+2]<<" | ";
        //std::cout<<adj[3*idx+ 0]<<" "<<adj[3*idx+ 1]<<" "<<adj[3*idx+ 2]<<" | "<<std::endl;
        idx++;
    }
	delete Tr;
	}

		
    // Transfer arrays to device.
    hipMemcpy(cu_r, r,                   2*tnumber*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cu_triangles, triangles,   3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_adj, adj,               3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_seed, seed,    		  tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_max, max,               tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_disconnect, disconnect, 3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_mesh, mesh,             3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_ind_poly, ind_poly,    tnumber*sizeof(int), hipMemcpyHostToDevice);
	
	//se consigue el indice de la malla i_mesh
	int i_mesh = 0;
	int *cu_i_mesh;
	hipMalloc((void**) &cu_i_mesh, sizeof(int));
	hipMemcpy(cu_i_mesh, &i_mesh, 1*sizeof(int), hipMemcpyHostToDevice);
	
	//se consigue el indice de ind_poly
	int i_ind_poly = 0;
	int *cu_i_ind_poly;
	hipMalloc((void**) &cu_i_ind_poly, sizeof(int));
	hipMemcpy(cu_i_ind_poly, &i_ind_poly, 1*sizeof(int), hipMemcpyHostToDevice);
	

	int enumber = 3*tnumber;

	//https://stackoverflow.com/questions/47822784/calculating-grid-and-block-dimensions-of-a-kernel
	int numThreads = 128;  // max register per block is 65536, 65536/512
	//int numBlocks  = (int)tnumber/numThreads;
	int numBlocks  = (tnumber + (numThreads-1))/numThreads;
	int numBlocks_edge  = (enumber + (numThreads-1))/numThreads;


	//Inicializar seeds y disconnect
	initialize_memory<<<numBlocks, numThreads>>>(cu_seed, cu_disconnect, tnumber);
	hipDeviceSynchronize();

	auto t1 = std::chrono::high_resolution_clock::now();
	auto tb_label =std::chrono::high_resolution_clock::now();	

	//Label phase
	//Etiquetar el más largo;
	std::cout<<"Inicia label longest"<<std::endl;
	label_longest_edges<<<numBlocks, numThreads>>>(cu_max, cu_r, cu_triangles, tnumber);
	hipDeviceSynchronize();
	
	//Encontrar un triangulo semilla asociado al arco terminal
	//get_seeds<<<numBlocks, numThreads>>>(cu_max, cu_triangles, cu_adj, cu_seed, tnumber);
	std::cout<<"inicia get seeds"<<std::endl;
	get_seeds<<<numBlocks_edge, numThreads>>>(cu_max, cu_triangles, cu_adj, cu_seed, enumber);
	hipDeviceSynchronize();
	//Etiquetar label frontier-edges
	//label_frontier_edges<<<numBlocks, numThreads>>>(cu_max, cu_disconnect, cu_triangles, cu_adj, tnumber);
	label_frontier_edges<<<numBlocks_edge, numThreads>>>(cu_max, cu_disconnect, cu_triangles, cu_adj, enumber);
	hipDeviceSynchronize();
	std::cout<<"terminado label frontier"<<std::endl;
	//Desconectar frontier-edges
	//disconnect_edges<<<numBlocks, numThreads>>>(cu_adj, cu_disconnect, tnumber);
	disconnect_edges<<<numBlocks_edge, numThreads>>>(cu_adj, cu_disconnect, enumber);
	hipDeviceSynchronize();
	std::cout<<"terminado disconnect"<<std::endl;
	auto te_label =std::chrono::high_resolution_clock::now();	


	//Se ordenan las semillas
	//hipMemcpy(adj, cu_adj,3*tnumber*sizeof(int), hipMemcpyDeviceToHost);
	//for (i = 0; i < tnumber; i++)
	//	std::cout<<adj[3*i+0]<<" "<<adj[3*i+1]<<" "<<adj[3*i+2]<<"\n";

	//hipMemcpy(seed, cu_seed,tnumber*sizeof(int), hipMemcpyDeviceToHost);
	//int num_region = 0;
	//for (i = 0; i < tnumber; i++)
	//{	
	//	if(seed[i] == TRUE){
	//		seed[num_region] = i;
	//		num_region++;
	//	}
	//}
	//for (i = 0; i < num_region; i++)
	//	std::cout<<seed[i]<<" ";
	//std::cout<<"\nregiones = "<<num_region<<std::endl;


	auto tb_travel = std::chrono::high_resolution_clock::now();
	generate_mesh<<<numBlocks, numThreads>>>(cu_triangles, cu_adj, cu_r, cu_seed, cu_mesh, tnumber, cu_i_mesh, cu_ind_poly, cu_i_ind_poly);
	std::cout<<"terminado mesh generation"<<std::endl;
	hipDeviceSynchronize();
	auto te_travel = std::chrono::high_resolution_clock::now();
	auto t2 = std::chrono::high_resolution_clock::now();
	hipMemcpy(&i_mesh, cu_i_mesh,sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&i_ind_poly, cu_i_ind_poly, sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(mesh, cu_mesh,3*tnumber*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(ind_poly, cu_ind_poly, tnumber*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(seed, cu_seed,tnumber*sizeof(int), hipMemcpyDeviceToHost);
	
	int num_region = 0;
	for (i = 0; i < tnumber; i++)
	{	
		if(seed[i] == TRUE){
			num_region++;
		}
	}
	
	write_geomview(r, triangles, pnumber, tnumber, i_mesh, mesh, seed, num_region, 0);

	std::cout << std::setprecision(3) << std::fixed;
    std::cout <<"pnumber tnumber num_reg tlabel talgorithm ttravel"<<std::endl;
	std::cout<<pnumber<<" "<<tnumber<<" "<<num_region;
	std::cout<<" "<<std::chrono::duration_cast<std::chrono::milliseconds>(te_label - tb_label).count();
	std::cout<<" "<<std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1 ).count();
	std::cout<<" "<<std::chrono::duration_cast<std::chrono::milliseconds>(te_travel - tb_travel ).count();


  	//imprimir polginos
	std::cout<<std::endl;
	int length_poly;
    i = 0;
    while(i < i_mesh){
        length_poly = mesh[i];
        std::cout<<"("<<i<<") "<<length_poly<<": ";
		i++;
        for(j=0; j < length_poly;j++){
            std::cout<< mesh[i]<<" ";
            i++;
        }
        std::cout<<std::endl;
    }

	for(i = 0; i < i_ind_poly; i++)	
		std::cout<< ind_poly[i]<<" ";
	std::cout<<std::endl;
	
	free(r);
	free(triangles);
	free(adj);
	free(seed );
	free(mesh);
	free(max);
	free(ind_poly);
	hipFree(cu_r);
	hipFree(cu_triangles);
	hipFree(cu_adj);
	hipFree(cu_seed);
	hipFree(cu_mesh);
	hipFree(cu_max);
	hipFree(cu_i_mesh);
	hipFree(cu_disconnect);
	hipFree(cu_ind_poly);
	return EXIT_SUCCESS;
}
    

